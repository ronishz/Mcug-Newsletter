
#include <hip/hip_runtime.h>
#include<stdio.h>
#define SIZE 10
#define BLOCKS 1
#define THREADS_PER_BLOCK 10

__global__ void oddevensort(int *in, int *out, int size)
{
	bool oddeven=true;
	__shared__ bool swappedodd;
	__shared__ bool swappedeven;
	int temp;
	swappedodd=true;
	swappedeven=true;

	while(true)
	{
		if(oddeven==true)
		{
			printf(" \n Swapping at odd locations ");
			__syncthreads();
			swappedodd=false;
			__syncthreads();

			int idx=threadIdx.x + blockIdx.x * blockDim.x;
			if(idx < (size / 2))
			{
				if(in[2 * idx] > in[2 * idx +1])
				{
					printf("\n Thread Id %d : is swapping %d <-> %d  \n Thread Id %d : [%d] <-> [%d] \n ", idx, in[2 * idx] ,  in[2 * idx + 1], idx, 2 * idx, (2 * idx +1));		

					temp = in[2 * idx];
					in [2 * idx]= in[2 * idx + 1];
					in [2 * idx + 1]=temp;
					swappedodd = true;
				}
			}

			__syncthreads();
		}

		else
		{
			//printf("Swapping at even locations \n ");
			__syncthreads();
			swappedeven=false;
			__syncthreads();

			int idx=threadIdx.x + blockIdx.x * blockDim.x;
			if(idx < (size / 2) - 1)
			{
				if(in[2 * idx + 1] > in[2 * idx +2])
				{
					printf("\n Thread Id %d : is swapping %d <-> %d  \n Thread Id %d : [%d] <-> [%d] \n ", idx, in[2 * idx + 1] ,  in[2 * idx + 2], idx, 2 * idx + 1, (2 * idx +2));		

					temp = in[2 * idx + 1];
					in [2 * idx + 1]= in[2 * idx + 2];
					in [2 * idx + 2] = temp;
					swappedeven=true;
				}
			}
			__syncthreads();
		}

		if(!(swappedodd || swappedeven ))
		break;
		oddeven = !oddeven;
	}

	__syncthreads();

	int idx =threadIdx.x;

	if(idx < size)
		out[idx] = in[idx];
}

int main(void)
{
	int *a, *a_sorted, i;
	int *d_a, *d_sorted;
	int size = sizeof(int)*SIZE;
	
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_sorted, size);

	a = (int*)malloc(size);
	a_sorted = (int*)malloc(size);

	hipMalloc((void**)&d_sorted,size);
        
	printf("\n Enter % d numbers to sort : \n",SIZE);
	for(i=0 ; i<SIZE; i++)
	{
		scanf("%d", &a[i]);
	}

	printf("\n Unsorted array is : \n");
	for(i=0 ; i<SIZE; i++)
	{
		printf("%d ", a[i]);
	}

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	oddevensort<<<BLOCKS,THREADS_PER_BLOCK>>>(d_a,d_sorted,SIZE);
	hipMemcpy(a,d_a,size,hipMemcpyDeviceToHost);
	hipMemcpy(a_sorted,d_sorted,size,hipMemcpyDeviceToHost);
	
	printf("\n \n Sorted array is : \n");
	for(i=0 ; i<SIZE; i++)
	{
		printf("%d ", a_sorted[i]);
	}
	
	printf("\n\n");

	free(a);
	free(a_sorted);

	hipFree(d_sorted);
	hipFree(d_a);
	
	return 0;
}
	
	
