#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#define SIZE 10

__global__ void DotProd(int *a, int *b, int *c)
{
	__shared__ int temp[SIZE];
	int x=threadIdx.x + blockDim.x * blockIdx.x;

	temp[threadIdx.x] = a[x] * b[x];
//	temp[threadIdx.y = a[y] * b[y];

	printf("%d:", temp[x]);
//	printf("%d:", temp[y]);
	
	printf("\n Thread ID : %d ",threadIdx.x);
	__syncthreads();

	if(x==0)
	{
		int i,sum=0;
		for(i=0;i<SIZE;i++)
		{
			sum += temp[i];
		//	atomicAdd(c,sum);
		}
		*c = sum;
		
		printf("\n\n Block Dimensional is : %d ",blockDim);
	        printf("\n\n Block ID : %d ",blockIdx.x);

		printf("\n \n Sum : %d" ,*c); 
	}
}
int main()
{
	int *a,*b,*c;
	int *d_a, *d_b,*d_c;
	int n = SIZE * sizeof(int);
	int i;

	a=(int*)malloc(n);
	b=(int*)malloc(n);
	c=(int*)malloc(n);

	for(i=0;i<SIZE;i++)
	{
		a[i]=i+1;
		b[i]=2*(i+1);
	}

	hipMalloc(&d_a,n);
	hipMalloc(&d_b,n);
	hipMalloc(&d_c, sizeof(int));

	hipMemcpy(d_a, a, n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n, hipMemcpyHostToDevice);
	printf("\n Launching kernel :- ");
	DotProd <<<1,SIZE>>> (d_a, d_b, d_c);

	hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("\n Dot Product is : %d \n\n" ,*c);
	
	free(a);
	free(b);
	free(c);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
